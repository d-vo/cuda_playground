#include "cudaResize.cuh"
#include <nppi_geometry_transforms.h>

#include <array>


bool
resizeYuv420(const unsigned char *src, int srcWidth, int srcHeight, unsigned char *dst, int dstWidth,
             int dstHeight, hipStream_t stream) {
    if (srcWidth % 2 != 0 or srcHeight % 2 != 0 or dstWidth % 2 != 0 or dstHeight % 2 != 0) {
        return false;
    }
    NppStreamContext nppStreamCtx;
    nppStreamCtx.hStream = stream;

    // Source
    const int srcUOffset = srcWidth * srcHeight;
    const int srcVOffset = srcWidth * srcHeight + srcWidth * srcHeight / 4;
    std::array<int, 3> srcWidths = {srcWidth, srcWidth / 2, srcWidth / 2};
    std::array<int, 3> srcHeights = {srcHeight, srcHeight / 2, srcHeight / 2};
    std::array<const unsigned char *, 3> srcPointers = {src, &src[srcUOffset], &src[srcVOffset]};

    // Destination
    const int dstUOffset = dstWidth * dstHeight;
    const int dstVOffset = dstWidth * dstHeight + dstWidth * dstHeight / 4;
    std::array<int, 3> dstWidths = {dstWidth, dstWidth / 2, dstWidth / 2};
    std::array<int, 3> dstHeights = {dstHeight, dstHeight / 2, dstHeight / 2};
    std::array<unsigned char *, 3> dstPointers = {dst, &dst[dstUOffset], &dst[dstVOffset]};

    NppStatus st;
    for (size_t i = 0; i < 3; ++i) {
        NppiSize srcSize = {srcWidths[i], srcHeights[i]};
        NppiRect srcRoi = {0, 0, srcSize.width, srcSize.height};
        NppiSize dstSize = {dstWidths[i], dstHeights[i]};
        NppiRect dstRoi = {0, 0, dstSize.width, dstSize.height};

        st = nppiResize_8u_C1R_Ctx(srcPointers[i], srcWidths[i], srcSize, srcRoi,
                                   dstPointers[i], dstWidths[i], dstSize, dstRoi, NPPI_INTER_LANCZOS,
                                   nppStreamCtx);
    }

    if (st != NPP_SUCCESS) {
        return false;
    }

    return true;
}


bool
resize(const unsigned char *src, int srcWidth, int srcHeight, unsigned char *dst, int dstWidth, int dstHeight,
       Format format, hipStream_t stream) {
    if (format == Format::YUV420) {
        return resizeYuv420(src, srcWidth, srcHeight, dst, dstWidth,
                            dstHeight, stream);

    }

    return false;
}
